#include "hip/hip_runtime.h"
#include "caffe2/core/context_gpu.h"
#include "yolo_op.h"
#include "caffe2/utils/math.h"

namespace caffe2 {
namespace {

static int entry_index(int batch, int location, int entry,int w,int h,int outputs,int classes)
{
    int n =   location / (w*h);
    int loc = location % (w*h);
    return batch*outputs + n*w*h*(4+classes+1) + entry*w*h + loc;
}

template <typename T>
__global__ void fillKernel(const int N, const T* x, T* y) {
  CUDA_1D_KERNEL_LOOP(i, N) {
    y[i] = x[i];
  }
}


template <typename T>
__global__ void SigmoidKernel(const int N, const T* x, T* y) {
  CUDA_1D_KERNEL_LOOP(i, N) {
    y[i] = 1. / (1. + exp(-x[i]));
  }
}

template <typename T>
__global__ void GenerateGridKernel(const int N, const int w,const int h,T* out1/*,T* out2*/) {
//    int index = 0;
  CUDA_1D_KERNEL_LOOP(i, N) {
    out1[i] = out1[i] + i%w;   
//    out2[i] = (out2[i] + index/h);
//    index++;
//    if(index == h*w)
//        index = 0;
  }
}

template <typename T>
__global__ void thtwKernel(const int N, T* x,T* y) {
  CUDA_1D_KERNEL_LOOP(i, N) {
          y[i] = exp(y[i])*x[i];
  }
}

}  // namespace


template<>
bool YoloOp<float,HIPContext>::RunOnDevice() {
    auto& X = Input(0);
    auto * Y = Output(0);
    auto * tx = Output(1);
    auto * ty = Output(2);
    auto * tw = Output(3);
    auto * th = Output(4);
    auto * det_conf = Output(5);
    auto * class_prob = Output(6);
    auto * tmp_blob = Output(7);
    Y->ResizeLike(X);
    Y->CopyFrom(X);
    int batch = Y->dim(0);
    int h = Y->dim(2);
    int w = Y->dim(3);
    this->h_=h;
    this->w_=w;
    this->batch_ = batch;
    for (int b = 0; b < batch; ++b){
        for(int n = 0; n < this->anchor_mask_.size(); ++n){
            int index = entry_index(b, n*w*h, 0,w,h,h*w*Y->dim(1),this->numclass_);
            SigmoidKernel<float><<<CAFFE_GET_BLOCKS(2*h*w), CAFFE_CUDA_NUM_THREADS,
                            0,context_.hip_stream()>>>(2*h*w,Y->template mutable_data<float>() + index ,
                                                                Y->template mutable_data<float>() + index);
            index = entry_index(b, n*w*h, 4,w,h,h*w*Y->dim(1),this->numclass_);
            SigmoidKernel<float><<<CAFFE_GET_BLOCKS((1+this->numclass_)*w*h), CAFFE_CUDA_NUM_THREADS,
                            0,context_.hip_stream()>>>((1+this->numclass_)*w*h, Y->template mutable_data<float>() + index
                                                                , Y->template mutable_data<float>() + index);
        }
    }
    tx->Resize(batch*this->anchor_mask_.size()*h*w);
    ty->ResizeLike(*tx);
    th->ResizeLike(*tx);
    tw->ResizeLike(*tx);
    tmp_blob->Resize(batch*this->anchor_mask_.size()*h*w);
    det_conf->ResizeLike(*tx);

    std::vector<int>shape;
    shape.push_back(batch);
    shape.push_back(this->anchor_mask_.size());
    shape.push_back(h);
    shape.push_back(w);
    tx->Reshape(shape);
    int index = 0;
    int step = h*w;
    class_prob->Resize(batch*this->anchor_mask_.size()*h*w*this->numclass_);
    for(int b = 0; b < batch; ++b){
        for(int n = 0; n < this->anchor_mask_.size(); ++n){
            fillKernel<float><<<CAFFE_GET_BLOCKS(h*w), CAFFE_CUDA_NUM_THREADS,
                            0,context_.hip_stream()>>>(h*w,Y->template mutable_data<float>() + index ,
                                                                tx->template mutable_data<float>() + step*n);

            fillKernel<float><<<CAFFE_GET_BLOCKS(h*w), CAFFE_CUDA_NUM_THREADS,
                            0,context_.hip_stream()>>>(h*w,Y->template mutable_data<float>() + index + h*w ,
                                                                ty->template mutable_data<float>() + step*n);

            fillKernel<float><<<CAFFE_GET_BLOCKS(h*w), CAFFE_CUDA_NUM_THREADS,
                            0,context_.hip_stream()>>>(h*w,Y->template mutable_data<float>() + index + 2*h*w ,
                                                                tw->template mutable_data<float>() + step*n);

            fillKernel<float><<<CAFFE_GET_BLOCKS(h*w), CAFFE_CUDA_NUM_THREADS,
                            0,context_.hip_stream()>>>(h*w,Y->template mutable_data<float>() + index + 3*h*w ,
                                                                th->template mutable_data<float>() + step*n);

            fillKernel<float><<<CAFFE_GET_BLOCKS(h*w), CAFFE_CUDA_NUM_THREADS,
                            0,context_.hip_stream()>>>(h*w,Y->template mutable_data<float>() + index + 4*h*w ,
                                                                det_conf->template mutable_data<float>() + step*n);

            fillKernel<float><<<CAFFE_GET_BLOCKS(h*w*this->numclass_), CAFFE_CUDA_NUM_THREADS,
                            0,context_.hip_stream()>>>(h*w*this->numclass_,Y->template mutable_data<float>() + index + 5*h*w ,
                                                                class_prob->template mutable_data<float>() + step*n*this->numclass_);
            index += h*w*(5+this->numclass_);
        }
    }

    std::vector<int>sp;
    sp.push_back(batch*this->anchor_mask_.size()*h*w);
    tx->Reshape(sp);
    GenerateGridKernel<float><<<CAFFE_GET_BLOCKS(h*w*batch*this->anchor_mask_.size()),CAFFE_CUDA_NUM_THREADS,
            0,context_.hip_stream()>>>(h*w*this->anchor_mask_.size()*batch,w,h,
                                        tx->template mutable_data<float>()/*,ty->template mutable_data<float>()*/);
    std::vector<float>ty_host;
    this->GetTensorToHost(ty,ty_host);
    for(int i =0;i < batch*h*w*this->anchor_mask_.size()/(h*w);i++){
        for(int j = 0;j < h*w;j ++){
            ty_host[j +i*h*w] = ty_host[j + i*h*w] + j/h;
        }
    }

    this->SetDeviceTensor(ty_host,ty);


    std::vector<float>aw,ah;
    for(int i = 0;i < this->anchor_mask_.size();i++){
        int ind = i*2;
        aw.push_back(this->masked_anchor_[ind]);
        ah.push_back(this->masked_anchor_[ind + 1]);
    }
    std::vector<float>anchor_w,anchor_h;
    tmp_blob->CopyFrom(*tx);
    for(int i = 0;i < h*w*batch*this->anchor_mask_.size();i ++){
        anchor_w.push_back(aw[i/(h*w*batch)]);
        anchor_h.push_back(ah[i/(h*w*batch)]);
    }
    this->SetDeviceTensor(anchor_w,tmp_blob);
    thtwKernel<float><<<CAFFE_GET_BLOCKS(h*w*batch*this->anchor_mask_.size()),CAFFE_CUDA_NUM_THREADS,
            0,context_.hip_stream()>>>(h*w*this->anchor_mask_.size()*batch,tmp_blob->template mutable_data<float>(),tw->template mutable_data<float>());
    this->SetDeviceTensor(anchor_h,tmp_blob);
    thtwKernel<float><<<CAFFE_GET_BLOCKS(h*w*batch*this->anchor_mask_.size()),CAFFE_CUDA_NUM_THREADS,
            0,context_.hip_stream()>>>(h*w*this->anchor_mask_.size()*batch,tmp_blob->template mutable_data<float>(),th->template mutable_data<float>());



    std::vector<int>psp;
    psp.push_back(batch*this->anchor_mask_.size());
    psp.push_back(this->numclass_);
    psp.push_back(h*w);
    class_prob->Reshape(psp);
    computeMaxConfIndex(class_prob);
    GetTensorToHost(tx,this->tx_);
    GetTensorToHost(ty,this->ty_);
    GetTensorToHost(th,this->th_);
    GetTensorToHost(tw,this->tw_);
    GetTensorToHost(det_conf,this->det_conf_);
    GetYoloBoxes();
    tx->Reshape(shape);
    return true;
}

template<>
bool YoloGradientOp<float,HIPContext>::RunOnDevice() {
    CAFFE_NOT_IMPLEMENTED;
}

REGISTER_HIP_OPERATOR(Yolo,YoloOp<float,HIPContext>);
REGISTER_HIP_OPERATOR(YoloGradient,YoloGradientOp<float,HIPContext>);
}  // namespace caffe2

